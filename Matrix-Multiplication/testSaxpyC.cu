
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <omp.h>
#include <unistd.h>

double wctime() 
{
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

__global__ void saxpy_par(int n, float a, float *x, float *y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i] = a * x[i] + y[i];
	}
}

float saxpy_check(int n, float a, float *x, float *y, float *z)  {
 // a, x, y == original data for saxpy
 // z = result found -- with which to compare.
 float s=0.0, t = 0.0;
 for (int i=0; i<n; i++) {
      y[i] += a * x[i] ;
       s += (y[i] - z[i])*(y[i] - z[i]);
       t += z[i]*z[i];
 }
  if (t == 0.0) return(-1);
    else
  return(sqrt(s/t));
}

int main() {
	//size of vectors
	int n = 8388608; //8*1024*1024
	size_t size = n * sizeof(float);
	
	//allocate vectors on CPU
	float *x , *y, *z;
	x = (float *)malloc(size);
	y = (float *)malloc(size);
	z = (float *)malloc(size);
	
	//allocate vectors on GPU
	//cudaMalloc( void** devPtr, size_t size )
	//cudaSuccess = 0
	float *x_GPU, *y_GPU;
	if (hipMalloc((void**) &x_GPU, size) != 0) {
		return -1;
	}
	if (hipMalloc((void**) &y_GPU, size) != 0) {
		return -1;
	}
	float a = 1.0;
	int NITER = 100;
	a = a/(float)NITER;
	
	//Initialize x and y with random numbers
	for (int i = 0; i < n; i++) {
		x[i] = (float)rand()/(float)rand();
		y[i] = (float)rand()/(float)rand(); 
	}

	int vecLen;
	for (vecLen = 2048; vecLen <= n; vecLen*=2) {
		//set grid and block dimensions
		dim3 dimGrid(vecLen/1024);
		dim3 dimBlock(1024);
		//call saxpy_par kernel NITER times
		double t1 = wctime(); //start time
		for (int iter = 0; iter < NITER; iter++) {
			//copy vectors to GPU
			hipMemcpy(x_GPU, x, vecLen * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(y_GPU, y, vecLen * sizeof(float), hipMemcpyHostToDevice);
			saxpy_par<<<dimGrid, dimBlock>>>(vecLen, a, x_GPU, y_GPU);
			//Copy result to CPU so it can be passed to saxpy_check
			hipMemcpy(z, y_GPU, vecLen * sizeof(float), hipMemcpyDeviceToHost);
		}
		double t2 = wctime(); //end time
		//Check error
		float error = saxpy_check(vecLen, a, x, y, z);
		//get performance stats
		//Perform a multiply and an add for each element in both arrays (2 operations)
		//This happens 
		float flops = (2 * vecLen * NITER)/(t2 - t1);
		printf("** vecLen = %d, Mflops = %.2f, err = %.2e\n", vecLen, flops*1e-6, error);
	}
	free(x);
	free(y);
	free(z);
	
	hipFree(x_GPU);
	hipFree(y_GPU);

	return 0;
}
